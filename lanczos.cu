#include "hip/hip_runtime.h"
/*!
    \file lanczos.cu
    \brief Controller code for general Lanczos diagonalization
*/

// Katharine Hyatt
// A set of functions to implement the Lanczos method for a generic Hamiltonian
// Based on the codes Lanczos_07.cpp and Lanczos07.h by Roger Melko
//-------------------------------------------------------------------------------

#include"lanczos.h"

/*Function lanczos: takes a hermitian matrix H, tridiagonalizes it, and finds the n smallest eigenvalues - this version only returns eigenvalues, not
 eigenvectors.
---------------------------------------------------------------------------------------------------------------------------------------------------
Input: howMany, the number of Hamiltonians to process
numElem - the number of nonzero elements per matrix
Hamiltonian - an array of Hamiltonians, each element being a custom struct containing the rows, cols, and vals in COO format as well as the dimensions
maxIter, the starting number of iterations we'll try
numEig, the number of eigenvalues we're interested in seeing
convReq, the convergence we'd like to see
---------------------------------------------------------------------------------------------------------------------------------------------------
Output: h_ordered, the array of the numEig smallest eigenvalues, ordered from smallest to largest
---------------------------------------------------------------------------------------------------------------------------------------------------
*/
__host__ void lanczos(const int howMany, const int* numElem, d_hamiltonian*& Hamiltonian, double**& groundstates, double**& eigenvalues, int maxIter, const int numEig, const double convReq)
{

    //----------Initializing CUBLAS and CUSPARSE libraries as well as storage on GPU----------------

    int* dim = (int*)malloc(howMany*sizeof(int));
    for(int i = 0; i < howMany; i++)
    {
        dim[i] = Hamiltonian[i].sectorDim;
    }

    /*! 

    First it is necessary to create handles, streams, and to initialize the two CUDA libraries which will be used:
	\verbatim
    */
    hipStream_t stream[howMany];

    hipblasStatus_t hipblasStatus_t[howMany];

    hipblasHandle_t linAlgHandle;
    hipblasStatus_t[0] = hipblasCreate(&linAlgHandle);

    if (hipblasStatus_t[0] != HIPBLAS_STATUS_SUCCESS)
    {
        std::cout<<"Initializing CUBLAS failed! Error: "<<hipblasStatus_t[0]<<std::endl;
    }

    hipsparseHandle_t sparseHandle;
    hipsparseStatus_t cusparseStatus[howMany];
    cusparseStatus[0] = hipsparseCreate(&sparseHandle);

    if (cusparseStatus[0] != HIPSPARSE_STATUS_SUCCESS)
    {
        std::cout<<"Failed to initialize CUSPARSE! Error: "<<cusparseStatus[0]<<std::endl;
    }
    /*!
    \endverbatim
    
    The function also transforms the Hamiltonian into CSR format so that CUSPARSE can use it for matrix-vector multiplications.
    \verbatim
    */
    hipsparseMatDescr_t H_descr[howMany];
    for(int i = 0; i<howMany; i++)
    {
        cusparseStatus[i] = hipsparseCreateMatDescr(&H_descr[i]);

        if (cusparseStatus[i] != HIPSPARSE_STATUS_SUCCESS)
        {
            std::cout<<"Error creating matrix description: "<<cusparseStatus[i]<<std::endl;
        }
        cusparseStatus[i] = hipsparseSetMatType(H_descr[i], HIPSPARSE_MATRIX_TYPE_GENERAL);
        if (cusparseStatus[i] != HIPSPARSE_STATUS_SUCCESS)
        {
            std::cout<<"Error setting matrix type: "<<cusparseStatus[i]<<std::endl;
        }
        cusparseStatus[i] = hipsparseSetMatIndexBase(H_descr[i], HIPSPARSE_INDEX_BASE_ZERO);
        if (cusparseStatus[i] != HIPSPARSE_STATUS_SUCCESS)
        {
            std::cout<<"Error setting matrix index base: "<<cusparseStatus[i]<<std::endl;
        }

    }
    hipError_t status[howMany];
    int** d_H_rowPtrs;
    d_H_rowPtrs = (int**)malloc(howMany*sizeof(int*));

    for(int i = 0; i < howMany; i++)
    {
        status[i] = hipStreamCreate(&stream[i]);
        if (status[i] != hipSuccess)
        {
            std::cout<<"Error creating streams: "<<hipGetErrorString(status[i])<<std::endl;
        }
        status[i] = hipMalloc(&d_H_rowPtrs[i], (dim[i] + 1)*sizeof(int));
        if (status[i] != hipSuccess)
        {
            std::cout<<"Error allocating d_H_rowPtrs: "<<hipGetErrorString(status[i])<<std::endl;
        }
    }

    //---------------Converting from COO to CSR format for Hamiltonians----------------
    //hipsparseHybMat_t hyb_Ham[howMany];
    for(int i = 0; i < howMany; i++)
    {
        /*cusparseStatus[i] = hipsparseCreateHybMat(&hyb_Ham[i]);
        if (cusparseStatus[i] != cusparseStatus_SUCCESS)
        {
        	std::cout<<"Error creating HYB matrix: "<<cusparseStatus[i]<<std::endl;
        }

        cout<<"Done creating HYB matrices"<<endl;*/

        cusparseStatus[i] = hipsparseSetStream(sparseHandle, stream[i]);
        if (cusparseStatus[i] != HIPSPARSE_STATUS_SUCCESS)
        {
            std::cout<<"Error switching streams: "<<cusparseStatus[i]<<std::endl;
        }



        status[i] = hipPeekAtLastError();
        if (status[i] != hipSuccess)
        {
            std::cout<<"Error synchronizing stream: "<<hipGetErrorString(status[i])<<std::endl;
        }
        cusparseStatus[i] = hipsparseXcoo2csr(sparseHandle, Hamiltonian[i].rows, numElem[i], dim[i], d_H_rowPtrs[i], HIPSPARSE_INDEX_BASE_ZERO);

        if (cusparseStatus[i] != HIPSPARSE_STATUS_SUCCESS)
        {
            std::cout<<"Error converting to CSR: "<<cusparseStatus[i]<<std::endl;
        }

        status[i] = hipPeekAtLastError();
        if (status[i] != hipSuccess)
        {
            std::cout<<"Error synchronizing stream: "<<hipGetErrorString(status[i])<<std::endl;
        }

        /*cusparseStatus[i] = hipsparseDcsr2hyb(sparseHandle, dim[i], dim[i], H_descr[i], Hamiltonian[i].vals, d_H_rowPtrs[i], Hamiltonian[i].cols, hyb_Ham[i], 0, HIPSPARSE_HYB_PARTITION_AUTO);

        	if (cusparseStatus[i] != cusparseStatus_SUCCESS)
        {
        	std::cout<<"Error converting to HYB: "<<cusparseStatus[i]<<std::endl;
        }*/

    }
    /*!
    \endverbatim
    status[0] = hipPeekAtLastError();
    if (status[0] != hipSuccess)
    {
        std::cout<<"Error before thread sync: "<<hipGetErrorString(status[0])<<std::endl;
    }
    */
    //----------------Create arrays to hold current Lanczos vectors----------
    vector< vector<double> > h_a(howMany);

    vector< vector<double> > h_b(howMany);
    //Making the "random" starting vector

    /*! 

    The function then sets up Lanczos diagonalization by initializing a random starting vector on the CPU, creating storage for the Lanczos vectors on the GPU, and copying this starting vector across.
    
    \verbatim
    */
    double** v0 = (double**)malloc(howMany*sizeof(double*));
    double** v1 = (double**)malloc(howMany*sizeof(double*));
    double** v2 = (double**)malloc(howMany*sizeof(double*));
    double*** lanczosStore = (double***)malloc(howMany*sizeof(double**));

    double** host_v0 = (double**)malloc(howMany*sizeof(double*));

    for(int i = 0; i < howMany; i++)
    {
        status[i] = hipMalloc(&v0[i], dim[i]*sizeof(double));
        if (status[i] != hipSuccess)
        {
            std::cout<<"Error creating storage for v0 on GPU: "<<hipGetErrorString(status[i])<<std::endl;
        }
        status[i] = hipMalloc(&v1[i], dim[i]*sizeof(double));
        if (status[i] != hipSuccess)
        {
            std::cout<<"Error creating storage for v1 on GPU: "<<hipGetErrorString(status[i])<<std::endl;
        }
        status[i] = hipMalloc(&v2[i], dim[i]*sizeof(double));
        if (status[i] != hipSuccess)
        {
            std::cout<<"Error creating storage for v2 on GPU: "<<hipGetErrorString(status[i])<<std::endl;
        }
        lanczosStore[i] = (double**)malloc(maxIter*sizeof(double*));
        host_v0[i] = (double*)malloc(dim[i]*sizeof(double));


        for(int j = 0; j<dim[i]; j++)
        {
            host_v0[i][j] = 0.;
            if (j%4 == 0) host_v0[i][j] = 1. ;
            else if (j%5 == 0) host_v0[i][j] = -2.;
            else if (j%7 == 0) host_v0[i][j] = 3.;
            else if (j%9 == 0) host_v0[i][j] = -4.;

        }

        status[i] = hipMalloc(&lanczosStore[i][0], dim[i]*sizeof(double));

        if (status[i] != hipSuccess)
        {
            std::cout<<"Error creating storage for v0 in lanczosStore: "<<hipGetErrorString(status[i])<<std::endl;
        }

        status[i] = hipMemcpyAsync(v0[i], host_v0[i], dim[i]*sizeof(double), hipMemcpyHostToDevice, stream[i]);
        if (status[i] != hipSuccess)
        {
            std::cout<<"Error copying v0 to the device: "<<hipGetErrorString(status[i])<<std::endl;
        }
    }
    /*!
    \endverbatim
    First, storage variables are created to hold the results of the CUBLAS functions.
    \verbatim
    */

    //--------------Create dummy variables for CUBLAS functions----------------

    double* normTemp = (double*)malloc(howMany*sizeof(double));
    double* alpha = (double*)malloc(howMany*sizeof(double));
    double* beta = (double*)malloc(howMany*sizeof(double));

    double* dotTemp = (double*)malloc(howMany*sizeof(double));
    double* axpyTemp = (double*)malloc(howMany*sizeof(double));
    
    double** y = (double**)malloc(howMany*sizeof(double*));
    /*!
    \endverbatim
    */
    //--------------Generate first Lanczos vector--------------------------

    for(int i = 0; i < howMany; i++)
    {
        hipblasSetStream(linAlgHandle, stream[i]);
        hipsparseSetStream(sparseHandle, stream[i]);
        /*! 
        Then the initial multiplication to generate the first Lanczos vector is performed. 
        \verbatim
        */     
       hipblasStatus_t[i] = hipblasDnrm2(linAlgHandle, dim[i], v0[i], 1, &normTemp[i]);

        normTemp[i] = 1./normTemp[i];

        hipblasStatus_t[i] = hipblasDscal(linAlgHandle, dim[i], &normTemp[i], v0[i], 1);

        alpha[i] = 1.;
        beta[i] = 0.;

        hipMemcpyAsync(lanczosStore[i][0], v0[i], dim[i]*sizeof(double), hipMemcpyDeviceToDevice, stream[i]);

        //-----------Apply Hamiltonian to V0--------------------
        cusparseStatus[i] = hipsparseDcsrmv(sparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, dim[i], dim[i], numElem[i], &alpha[i], H_descr[i], Hamiltonian[i].vals, d_H_rowPtrs[i], Hamiltonian[i].cols, v0[i], &beta[i], v1[i]); // the Hamiltonian is applied here

        /*!
        \endverbatim
        */

        if (cusparseStatus[i] != HIPSPARSE_STATUS_SUCCESS)
        {
            std::cout<<"Getting V1 = H*V0 failed! Error: ";
            std::cout<<cusparseStatus[i]<<std::endl;
        }
        //hipStreamSynchronize(stream[i]);
        if (hipPeekAtLastError() != 0 )
        {
            std::cout<<"Getting V1  = H*V0 failed! Error: ";
            std::cout<<hipGetErrorString(hipPeekAtLastError())<<std::endl;
        }


    }

    for(int i = 0; i < howMany; i++)
    {
        hipblasSetStream(linAlgHandle, stream[i]);
        dotTemp[i] = 0.;

        hipblasStatus_t[i] = hipblasDdot(linAlgHandle, dim[i], v1[i], 1, v0[i], 1, &dotTemp[i]);
        
        h_a[i].push_back(dotTemp[i]);
        h_b[i].push_back(0.);

        if (hipblasStatus_t[i] != HIPBLAS_STATUS_SUCCESS)
        {
            std::cout<<"Getting d_a[0] failed! Error: ";
            std::cout<<hipblasStatus_t[i]<<std::endl;
        }

        

        //hipStreamSynchronize(stream[i]);
        if (hipblasStatus_t[i] != HIPBLAS_STATUS_SUCCESS)
        {
            std::cout<<"Getting h_a[0] failed! Error: ";
            std::cout<<hipblasStatus_t[i]<<std::endl;
        }

        

        if (status[i] != hipSuccess)
        {
            std::cout<<"Memory allocation of y dummy vector failed! Error:";
            std::cout<<hipGetErrorString( status[i] )<<std::endl;
        }

        status[i] = hipMalloc(&y[i], dim[i]*sizeof(double));

        
        /*!
        The new vector must be rescaled and stored before Lanczos iteration can begin. 
        \verbatim
        */

        hipblasStatus_t[i] = hipblasDscal(linAlgHandle, dim[i], &beta[i], y[i], 1);
        //hipStreamSynchronize(stream[i]);

        axpyTemp[i] = -1*h_a[i][0];

        hipblasStatus_t[i] = hipblasDaxpy(linAlgHandle, 0, &axpyTemp[i], v0[i], 1, v1[i], 1);
        //hipStreamSynchronize(stream[i]);

        if (hipblasStatus_t[i] != HIPBLAS_STATUS_SUCCESS)
        {
            std::cout<<"V1 = V1 - alpha*V0 failed! Error: ";
            std::cout<<hipblasStatus_t[i]<<std::endl;
        }
        if (hipPeekAtLastError() != 0 )
        {
            std::cout<<"Getting V1  = V1 - a*V0 failed! Error: ";
            std::cout<<hipGetErrorString(hipPeekAtLastError())<<std::endl;
        }

        //---------Normalize V1 and copy it to Lanczos storage-----------

        normTemp[i] = 0.;
        hipblasStatus_t[i] = hipblasDnrm2(linAlgHandle, dim[i], v1[i], 1, &normTemp[i]); //this is slow for some reason

        //hipStreamSynchronize(stream[i]);

        if (hipblasStatus_t[i] != HIPBLAS_STATUS_SUCCESS)
        {
            std::cout<<"Getting the norm of v1 failed! Error: ";
            std::cout<<hipblasStatus_t[i]<<std::endl;
        }

        if (hipPeekAtLastError() != 0 )
        {
            std::cout<<"Getting nrm(V1) failed! Error: ";
            std::cout<<hipGetErrorString(hipPeekAtLastError())<<std::endl;
        }

    }

    double* gamma = (double*)malloc(howMany*sizeof(double));

    for(int i = 0; i < howMany; i++)
    {
        hipblasSetStream(linAlgHandle, stream[i]);

        h_b[i].push_back(normTemp[i]);

        normTemp[i] = 1./normTemp[i];
        gamma[i] = 1./h_b[i][1]; //alpha = 1/beta in v1 = v1 - alpha*v0
        hipblasStatus_t[i] = hipblasDscal(linAlgHandle, dim[i], &normTemp[i], v1[i], 1);


        if (hipblasStatus_t[i] != HIPBLAS_STATUS_SUCCESS)
        {
            std::cout<<"Normalizing v1 failed! Error: ";
            std::cout<<hipblasStatus_t[i]<<std::endl;
        }


        if (hipPeekAtLastError() != 0 )
        {
            std::cout<<"Normalizing V1 failed! Error: ";
            std::cout<<hipGetErrorString(hipPeekAtLastError())<<std::endl;
        }
        hipMalloc(&lanczosStore[i][1], dim[i]*sizeof(double));
        hipMemcpyAsync(lanczosStore[i][1], v1[i], dim[i]*sizeof(double), hipMemcpyDeviceToDevice, stream[i]);
    }
    /*!    
    \endverbatim
    */

    /*! 
    Storage space for the tridiagonal matrix is created and flags are initialized to track progress:

	\verbatim
    */
    
    double* gsEnergy = (double*)malloc(howMany*sizeof(double));

    double* eigTemp = (double*)malloc(howMany*sizeof(double));

    int* returned = (int*)malloc(howMany*sizeof(int));

    int* iter = (int*)malloc(howMany*sizeof(int));

    bool* doneFlag = (bool*)malloc(howMany*sizeof(bool));

    double** h_H_eigen = (double**)malloc(howMany*sizeof(double*));
    double** d_H_eigen = (double**)malloc(howMany*sizeof(double*));

    double** h_diag = (double**)malloc(howMany*sizeof(double*));
    double** h_offdia = (double**)malloc(howMany*sizeof(double*));

    vector< vector < double > > h_ordered(howMany);
    /*!
    \endverbatim
    */
    /*! 
    The flags and storage are initialized for the interations
    \verbatim
    */
    for(int i = 0; i<howMany; i++)
    {
        gsEnergy[i] = 1.;
        eigTemp[i] = 0.;
        iter[i] = 0;
        doneFlag[i] = false;
        h_ordered[i].resize(numEig, 0);
        h_H_eigen[i] = (double*)malloc(maxIter*maxIter*sizeof(double));
        hipMalloc(&d_H_eigen[i], maxIter*maxIter*sizeof(double));
        h_diag[i] = (double*)malloc(h_a[i].size()*sizeof(double));
        h_offdia[i] = (double*)malloc(h_b[i].size()*sizeof(double));
    }
    /*!
    \endverbatim
    */

    //---------Begin Lanczos iteration-----------------------------

    bool allDone = false;

    while( !allDone )
    {
        allDone = true;
        for(int i = 0; i < howMany; i++)
        {
            hipblasSetStream(linAlgHandle, stream[i]);
            hipsparseSetStream(sparseHandle, stream[i]);
            hipStreamSynchronize(stream[i]);

            /*! 
            If the current diagonalization is not complete, multiply H*V1 to get a new V2
            \verbatim
            */
            if (!doneFlag[i])
            {
                iter[i]++;
                
                cusparseStatus[i] = hipsparseDcsrmv(sparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, dim[i], dim[i], numElem[i], &alpha[i], H_descr[i], Hamiltonian[i].vals, d_H_rowPtrs[i], Hamiltonian[i].cols, v1[i], &beta[i], v2[i]);
                
                if( cusparseStatus[i] != 0)
                {
                    cout<<"Error applying H to V1 in "<<iter[i]<<"th iteration"<<endl;
                }
                //cusparseStatus[i] = hipsparseDhybmv(sparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha[i], H_descr[i], hyb_Ham[i], v1[i], &beta[i], v2[i]); // the Hamiltonian is applied here, in this gross expression
            }
            /*!
            \endverbatim
            */
        }
        for(int i = 0; i < howMany; i++)
        {
            hipblasSetStream(linAlgHandle, stream[i]);
            if (!doneFlag[i])
            {
                if (cusparseStatus[i] != HIPSPARSE_STATUS_SUCCESS)
                {
                    std::cout<<"Error applying the Hamiltonian in "<<iter[i]<<"th iteration!";
                    std::cout<<"Error: "<<cusparseStatus[i]<<std::endl;
                }

                hipblasStatus_t[i] = hipblasDdot(linAlgHandle, dim[i], v1[i], 1, v2[i], 1, &dotTemp[i]);
                //hipStreamSynchronize(stream[i]);

                h_a[i].push_back(dotTemp[i]);


                if (hipblasStatus_t[i] != HIPBLAS_STATUS_SUCCESS)
                {
                    std::cout<<"Error getting v1 * v2 in "<<iter[i]<<"th iteration! Error: ";
                    std::cout<<hipblasStatus_t[i]<<std::endl;
                }

                axpyTemp[i] = -1.*h_b[i][iter[i]];

                hipblasStatus_t[i] = hipblasDaxpy( linAlgHandle, dim[i], &axpyTemp[i], v0[i], 1, v2[i], 1);
                if (hipblasStatus_t[i] != HIPBLAS_STATUS_SUCCESS)
                {
                    std::cout<<"Error getting (d_b/d_a)*v0 + v1 in "<<iter[i]<<"th iteration!";
                    std::cout<<"Error: "<<hipblasStatus_t[i]<<std::endl;
                }

            }
        }

        //--------Find next set of elements in Lanczos Hamiltonian----

        for(int i = 0; i < howMany; i++)
        {
            hipblasSetStream(linAlgHandle, stream[i]);
            //hipStreamSynchronize(stream[i]);
            if (!doneFlag[i])
            {
                /*! 
                Similarly to setting up V1, V2 must be rescaled
                \verbatim
                */
                axpyTemp[i] = -1.*h_a[i][iter[i]];
                hipblasStatus_t[i] = hipblasDaxpy( linAlgHandle, dim[i], &axpyTemp[i], v1[i], 1, v2[i], 1);
                if (hipblasStatus_t[i] != HIPBLAS_STATUS_SUCCESS)
                {
                    std::cout<<"Error getting v2 + d_a*v1 in "<<iter[i]<<"th iteration! Error: ";
                    std::cout<<hipblasStatus_t[i]<<std::endl;
                }

                hipblasStatus_t[i] = hipblasDnrm2( linAlgHandle, dim[i], v2[i], 1, &normTemp[i]);
                
                if (hipblasStatus_t[i] != HIPBLAS_STATUS_SUCCESS)
                {
                    std::cout<<"Error getting norm of v2 in "<<iter[i]<<"th iteration! Error: ";
                    std::cout<<hipblasStatus_t[i]<<std::endl;
                }


                h_b[i].push_back(normTemp[i]);
                gamma[i] = 1./normTemp[i];
                /*!
                \endverbatim
                */
                hipblasStatus_t[i] = hipblasDscal(linAlgHandle, dim[i], &gamma[i], v2[i], 1);
                if (hipblasStatus_t[i] != HIPBLAS_STATUS_SUCCESS)
                {
                    std::cout<<"Error getting 1/d_b * v2 in "<<iter[i]<<"th iteration! Error: ";
                    std::cout<<hipblasStatus_t[i]<<std::endl;
                }
            }
        }


        for(int i = 0; i < howMany; i++)
        {
            hipblasSetStream(linAlgHandle, stream[i]);
            //status[i] = hipStreamSynchronize(stream[i]);

            if (status[i] != hipSuccess)
            {
                std::cout<<"Error syncing before copying v1 to v0: "<<hipGetErrorString(status[i])<<std::endl;
            }

            if (!doneFlag[i])
            {
                /*!
                Reorthogonalization is performed on v2 to ensure that the excited states do not collapse into the groundstate
                \verbatim
                */
                for(int j = 0; j < iter[i] + 1; j++)
                {
                    hipblasDdot(linAlgHandle, dim[i], v2[i], 1, lanczosStore[i][j], 1, &dotTemp[i]);
                    dotTemp[i] *= -1.;
                    hipblasDaxpy(linAlgHandle, dim[i],  &dotTemp[i], lanczosStore[i][j], 1, v2[i], 1);
                    dotTemp[i] = 1. - dotTemp[i]*dotTemp[i];
                    hipblasDscal(linAlgHandle, dim[i], &dotTemp[i], v2[i], 1);

                }
                /*!
                \endverbatim
                The vectors are copied down one and stored to prepare for the next iteration
                \verbatim
                */

                status[i] = hipMemcpyAsync(v0[i], v1[i], dim[i]*sizeof(double), hipMemcpyDeviceToDevice, stream[i]);
                if (status[i] != hipSuccess)
                {
                    std::cout<<"Error copying v1 to v0: "<<hipGetErrorString(status[i])<<std::endl;
                }
                status[i] = hipMemcpyAsync(v1[i], v2[i], dim[i]*sizeof(double), hipMemcpyDeviceToDevice, stream[i]);
                if (status[i] != hipSuccess)
                {
                    std::cout<<"Error copying v2 to v1: "<<hipGetErrorString(status[i])<<std::endl;
                }
             
                status[i] = hipMalloc(&lanczosStore[i][iter[i] + 1], dim[i]*sizeof(double));
                status[i] = hipMemcpyAsync(lanczosStore[i][iter[i] + 1], v2[i], dim[i]*sizeof(double), hipMemcpyDeviceToDevice, stream[i]);
                /*!
                \endverbatim
                */
            }

        }

        for(int i = 0; i < howMany; i++)
        {
            if (!doneFlag[i] && iter[i] > 5)
            {
                //---Copy Lanczos matrix information for diagonalization-----
                free(h_diag[i]);
                free(h_offdia[i]);
                h_diag[i] = (double*)malloc(h_a[i].size()*sizeof(double));
                h_offdia[i] = (double*)malloc(h_b[i].size()*sizeof(double));

                h_diag[i][0] = h_a[i][0];
                for (int ii=1; ii<=iter[i]; ii++)
                {
                    h_diag[i][ii] = h_a[i][ii];
                    h_offdia[i][ii] = h_b[i][ii];
                    h_offdia[i][ii-1] = h_offdia[i][ii];
                }
                h_offdia[i][iter[i]] = 0;

                //hipStreamSynchronize(stream[i]);

                //---------Diagonalize Lanczos matrix and check for convergence------------------
                returned[i] = tqli(h_diag[i], h_offdia[i], iter[i] + 1, maxIter, h_H_eigen[i]);
                status[i] = hipPeekAtLastError();
                if( status[i] != hipSuccess)
                {
                    cout<<"Error in identity! Error: "<<hipGetErrorString(status[i])<<endl;
                }
                //cout<<"Done tqli in "<<iter[i]<<"th iteration"<<endl;
                hipMemcpyAsync(d_H_eigen[i], h_H_eigen[i], maxIter*maxIter*sizeof(double), hipMemcpyHostToDevice, stream[i]);

                std::sort(h_diag[i], h_diag[i] + h_a[i].size());
                for (int j = 0; j < numEig; j++)
                {
                    h_ordered[i][j] = h_diag[i][j];
                    //cout<<h_ordered[i][j]<<" ";
                }
                //cout<<endl;

                gsEnergy[i] = h_ordered[i][numEig - 1];
                doneFlag[i] = (fabs(gsEnergy[i] - eigTemp[i]) < convReq);// && iter[i] > 10;// ? (iter[i] > 10) : false;
                //doneFlag[i] = iter[i] == maxIter - 2;
                eigTemp[i] = h_ordered[i][numEig - 1];

                if (iter[i] == maxIter - 2) // have to use this or d_b will overflow
                {
                    //this stuff here is used to resize the main arrays in the case that we aren't converging quickly enough
                    h_a[i].resize(2*maxIter);
                    h_b[i].resize(2*maxIter);
                    maxIter *= 2;
                }
            }
        }
        allDone = true;
        for(int i = 0; i< howMany; i++)
        {
            allDone = (allDone && doneFlag[i]);
        }
    }


    //-------------Get groundstates------------------------------------------

    for( int i = 0; i < howMany; i++)
    {
        //hipStreamSynchronize(stream[i]);
        //GetGroundstate<<<dim[i]/512 + 1, 512, 0, stream[i]>>>(groundstates[i], lanczosStore[i], d_H_eigen[i], iter[i], dim[i]);
    }

    //--------------Free arrays to prevent memory leaks------------------------
    for(int i = 0; i < howMany; i++)
    {
        for(int j = 0; j < numEig; j++)
        {
            std::cout<<std::setprecision(12)<<h_ordered[i][j]<<" ";
        }
        std::cout<<std::endl;

        for(int j = 0; j < iter[i]; j++)
        {
            hipFree(lanczosStore[i][j]);
        }
        free(lanczosStore[i]);
        hipFree(d_H_rowPtrs[i]);
        hipFree(v0[i]);
        hipFree(v1[i]);
        hipFree(v2[i]);
        hipFree(y[i]);
        free(h_H_eigen[i]);
        hipFree(d_H_eigen[i]);
        free(host_v0[i]);
        free(h_diag[i]);
        free(h_offdia[i]);
        //hipsparseDestroyHybMat(hyb_Ham[i]);
    }

    free(gsEnergy);
    free(eigTemp);
    free(alpha);
    free(beta);
    free(returned);
    free(iter);
    free(doneFlag);
    free(h_H_eigen);
    free(d_H_eigen);
    free(gamma);
    free(y);
    free(normTemp);
    free(axpyTemp);
    free(dotTemp);
    free(host_v0);
    free(v0);
    free(v1);
    free(v2);
    free(h_diag);
    free(h_offdia);
    free(lanczosStore);
    //free(dim);
    free(d_H_rowPtrs);

    hipblasStatus_t[0] = hipblasDestroy(linAlgHandle);

    //----------Output groundstate to file to check for correctness------

    double* host_groundstate = (double*)malloc(dim[0]*sizeof(double));
    /*std::ofstream fout;
    fout.open("lanczos.log");
    hipMemcpy(host_groundstate, groundstates[0], dim[0]*sizeof(double), hipMemcpyDeviceToHost);
    for(int i = 0; i < dim[0] ; i++)
    {
        fout<<host_groundstate[i]<<std::endl;
    }

    fout.close();*/
    free(host_groundstate);
    free(dim);
    if (hipblasStatus_t[0] != HIPBLAS_STATUS_SUCCESS)
    {
        printf("CUBLAS failed to shut down properly! \n");
    }

    cusparseStatus[0] = hipsparseDestroy(sparseHandle);

    if (cusparseStatus[0] != HIPSPARSE_STATUS_SUCCESS)
    {
        printf("CUSPARSE failed to release handle! \n");
    }


    /*if (iter == 1) {
    	std::ofstream fout;
    	fout.open("lanczos.log");
    	//fout<<normTemp<<std::endl;
    	fout<<std::endl;
    	//int* h_H_vals = (int*)malloc((dim+1)*sizeof(int));
    	hipMemcpy(host_v0, v2, dim*sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);
    	for(int i = 0; i < dim ; i++){
    		fout<<host_v0[i].x<<std::endl;
    	}

    	fout.close();
    }*/


}
// things left to do:
// write a thing (separate file) to call routines to find expectation values, should be faster on GPU
// make the tqli thing better!

int tqli(double* d, double* e, int n, int maxIter, double *z)

{

    int m,l,iter,i,k;
    double s,r,p,g,f,dd,c,b;

    for (l=0; l<n; l++)
    {
        iter=0;
        do
        {
            for (m=l; m<n-1; m++)
            {
                dd=fabs(d[m])+fabs(d[m+1]);
                if (fabs(e[m])+dd == dd) break;
            }
            if (m!=l)
            {
                if (iter++ == 60)
                {
                    std::cout <<"Too many iterations in tqli() \n";
                    return 0;
                }
                g=(d[l+1]-d[l])/(2.0*e[l]);
                r=sqrt((g*g)+1.0);
                g=d[m]-d[l]+e[l]/(g+SIGN(r,g));
                s=c=1.0;
                p=0.0;
                for (i=m-1; i>=l; i--)
                {
                    f=s*e[i];
                    b=c*e[i];
                    if (fabs(f) >= fabs(g))
                    {
                        c=g/f;
                        r=sqrt((c*c)+1.0);
                        e[i + 1]=f*r;
                        c *= (s=1.0/r);
                    }
                    else
                    {
                        s=f/g;
                        r=sqrt((s*s)+1.0);
                        e[i+1]=g*r;
                        s *= (c=1.0/r);
                    }
                    g=d[i+1]-p;
                    r=(d[i]-g)*s+2.0*c*b;
                    p=s*r;
                    d[i+1]=g+p;
                    g=c*r-b;
                    /*EVECTS*/


                    for (k=0; k<n; k++)
                    {
                        f=z[k * n + i+1];
                        z[k*n + i+1]=s*z[k*n + i]+c*f;
                        z[k*n + i ]=c*z[k*n+i]-s*f;
                    }

                }
                d[l]=d[l]-p;
                e[l]=g;
                e[m]=0.0;
            }
        }
        while (m!=l);
    }
    return 1;
}


double pythag(double a, double b)
{
    double absa, absb;
    absa=fabs(a);
    absb=fabs(b);
    if (absa > absb) return absa*sqrt(1.0+(absb/absa)*(absb/absa));
    else return (absb == 0.0 ? 0.0 : absb*sqrt(1.0+(absa/absb)*(absa/absb)));
}

__global__ void GetGroundstate(double* groundstates, double** lanczosStore, double* H_eigen, int mat_dim, int vec_dim)
{

    int element = blockIdx.x*blockDim.x + threadIdx.x;

    if ( element < vec_dim )
    {
        groundstates[element] = H_eigen[0]*lanczosStore[0][element];
        for (int lancIter = 1; lancIter < mat_dim; lancIter++)
        {
            groundstates[element] += H_eigen[lancIter]*lanczosStore[lancIter][element];
        }
    }
};
